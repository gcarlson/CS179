#include "hip/hip_runtime.h"

/* 
Based off work by Nelson, et al.
Brigham Young University (2010)

Adapted by Kevin Yuh (2015)
*/


#include <stdio.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hipfft/hipfft.h>

#define PI 3.14159265358979

texture<float, 2, hipReadModeElementType> texreference;

/* Check errors on CUDA runtime functions */
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess) 
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        exit(code);
    }
}



/* Check errors on cuFFT functions */
void gpuFFTchk(int errval){
    if (errval != HIPFFT_SUCCESS){
        printf("Failed FFT call, error code %d\n", errval);
    }
}



/* Check errors on CUDA kernel calls */
void checkCUDAKernelError()
{
    hipError_t err = hipGetLastError();
    if  (hipSuccess != err){
        fprintf(stderr, "Error %s\n", hipGetErrorString(err));
    } else {
        fprintf(stderr, "No kernel error detected\n");
    }

}


/* Kernel for ramp filtering */
__global__
void
cudaScaleKernel(hipfftComplex *sinogram_dev, int nAngles, int sinogram_width) {

    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    while (index < sinogram_width * nAngles) {
        // Scale highest frequencies by more
        sinogram_dev[index].x *= (1 - fabs((index % sinogram_width) * 2.0 
            / (sinogram_width - 1.0) - 1));
        sinogram_dev[index].y *= (1 - fabs((index % sinogram_width) * 2.0 
            / (sinogram_width - 1.0) - 1));

        index += blockDim.x * gridDim.x;
    }
}

/* Kernel for copying complex results to floats */
__global__
void
cudaMoveKernel(hipfftComplex *sinogram_dev, float *sinogram_dev_float,
    int nAngles, int sinogram_width) {

    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    while (index < sinogram_width * nAngles) {
        sinogram_dev_float[index] = sinogram_dev[index].x;
        index += blockDim.x * gridDim.x;
    }
}

/* Kernel to perform backprojection */
__global__
void
cudaBackProjectKernel(float *sinogram_dev_float, int width, int height,
    int sinogram_width, float *dev_output, int nAngles) {

    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    float m, q, d, t, xi, yi, xo, yo;
    while (index < width * height) {
        xo = index % width - width / 2.0;
        yo = height / 2.0 - (index + 0.0) / width;
        dev_output[index] = 0;
        // Handle edge cases (otherwise will divide by zero)
        for (int i = 0; i < nAngles; i++) {
            t = (PI * i) / nAngles;
            if (i == 0)
                d = xo;
            else if (2 * i == nAngles)
                d = yo;
            else {  
	    m = 0 - cos(t) / sin(t);
	    q = -1.0 / m;
	    xi = (yo - m * xo) / (q - m);
	    yi = q * xi;
	    d = sqrt(xi * xi + yi * yi);
	    if ((q > 0 && xi < 0) || (q < 0 && xi > 0))
		d = 0 - d;
            }
            dev_output[index] += tex2D(texreference, i,
                (int) (d + sinogram_width / 2.0));
            //dev_output[index] += sinogram_dev_float[i * sinogram_width + 
            //    (int) (d + sinogram_width / 2.0)];
        }
        index += blockDim.x * gridDim.x;
    }
}
  



int main(int argc, char** argv){

    if (argc != 7){
        fprintf(stderr, "Incorrect number of arguments.\n\n");
        fprintf(stderr, "\nArguments: \n \
        < Sinogram filename > \n \
        < Width or height of original image, whichever is larger > \n \
        < Number of angles in sinogram >\n \
        < threads per block >\n \
        < number of blocks >\n \
        < output filename >\n");
        exit(EXIT_FAILURE);
    }


    /********** Parameters **********/

    int width = atoi(argv[2]);
    int height = width;
    int sinogram_width = (int)ceilf( height * sqrt(2) );

    int nAngles = atoi(argv[3]);


    int threadsPerBlock = atoi(argv[4]);
    int nBlocks = atoi(argv[5]);


    /********** Data storage *********/


    // GPU DATA STORAGE
    hipfftComplex *dev_sinogram_cmplx;
    float *dev_sinogram_float; 
    float* output_dev;  // Image storage


    hipfftComplex *sinogram_host;

    size_t size_result = width*height*sizeof(float);
    float *output_host = (float *)malloc(size_result);




    /*********** Set up IO, Read in data ************/

    sinogram_host = (hipfftComplex *)malloc(  sinogram_width*nAngles*sizeof(hipfftComplex) );

    FILE *dataFile = fopen(argv[1],"r");
    if (dataFile == NULL){
        fprintf(stderr, "Sinogram file missing\n");
        exit(EXIT_FAILURE);
    }

    FILE *outputFile = fopen(argv[6], "w");
    if (outputFile == NULL){
        fprintf(stderr, "Output file cannot be written\n");
        exit(EXIT_FAILURE);
    }

    int j, i;

    for(i = 0; i < nAngles * sinogram_width; i++){
        fscanf(dataFile,"%f",&sinogram_host[i].x);
        sinogram_host[i].y = 0;
    }

    fclose(dataFile);


    /*********** Assignment starts here *********/

    /* Allocate memory for all GPU storage above, copy input sinogram
    over to dev_sinogram_cmplx. */
    hipMalloc(&dev_sinogram_cmplx, 
        nAngles * sinogram_width * sizeof(hipfftComplex));
    hipMalloc(&dev_sinogram_float, nAngles * sinogram_width * sizeof(float));
    hipMalloc(&output_dev, width * height * sizeof(float));  // Image storage

    hipMemcpy(dev_sinogram_cmplx, sinogram_host, 
        nAngles * sinogram_width * sizeof(hipfftComplex), hipMemcpyHostToDevice);
   

    /* The high-pass filter:
        - Use cuFFT for the forward FFT
        - Create your own kernel for the frequency scaling.
        - Use cuFFT for the inverse FFT
        - extract real components to floats
        - Free the original sinogram (dev_sinogram_cmplx)

        Note: If you want to deal with real-to-complex and complex-to-real
        transforms in cuFFT, you'll have to slightly change our code above.
    */
    hipfftHandle plan;
    // Use a batched FFT to transform each sinogram at once
    hipfftPlan1d(&plan, sinogram_width, HIPFFT_C2C, nAngles);

    hipfftExecC2C(plan, dev_sinogram_cmplx, dev_sinogram_cmplx, HIPFFT_FORWARD);
    cudaScaleKernel<<<nBlocks, threadsPerBlock>>>
        (dev_sinogram_cmplx, nAngles, sinogram_width);
    
    checkCUDAKernelError();
    
    hipfftExecC2C(plan, dev_sinogram_cmplx, dev_sinogram_cmplx, HIPFFT_BACKWARD);
    hipfftDestroy(plan);
    
    cudaMoveKernel<<<nBlocks, threadsPerBlock>>>
    	(dev_sinogram_cmplx, dev_sinogram_float, nAngles, sinogram_width);
    checkCUDAKernelError();

    hipArray* carray;
    hipChannelFormatDesc channel;
    channel = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);

    hipMallocArray(&carray, &channel, sinogram_width, nAngles);
    hipMemcpyToArray(carray, 0, 0, dev_sinogram_float, 
        nAngles * sinogram_width * sizeof(float), hipMemcpyDeviceToDevice);

    texreference.filterMode = hipFilterModeLinear;
    texreference.addressMode[0] = hipAddressModeClamp;
    texreference.addressMode[1] = hipAddressModeClamp;
    
    hipBindTextureToArray(texreference, carray);
    

    /* Backprojection.
        - Create your own kernel to accelerate backprojection.
        - Copy the reconstructed image back to output_host.
        - Free all remaining memory on the GPU.
    */
    cudaBackProjectKernel<<<nBlocks, threadsPerBlock>>>
        (dev_sinogram_float, width, height, sinogram_width, 
            output_dev, nAngles);
    checkCUDAKernelError();
           
    hipMemcpy(output_host, output_dev, width * height * sizeof(float),
        hipMemcpyDeviceToHost);    
    
    hipFree(dev_sinogram_cmplx);
    hipFree(dev_sinogram_float);
    hipFree(output_dev);    

    /* Export image data. */

    for(j = 0; j < width; j++){
        for(i = 0; i < height; i++){
            fprintf(outputFile, "%e ",output_host[j*width + i]);
        }
        fprintf(outputFile, "\n");
    }


    /* Cleanup: Free host memory, close files. */

    free(sinogram_host);
    free(output_host);

    fclose(outputFile);

    return 0;
}



